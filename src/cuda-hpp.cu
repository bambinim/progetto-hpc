#include "hip/hip_runtime.h"
/****************************************************************************
 *
 * hpp.c - Serial implementaiton of the HPP model
 *
 * Copyright (C) 2021 by Moreno Marzolla <moreno.marzolla(at)unibo.it>
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *    http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 * --------------------------------------------------------------------------
 *
 * Compile with
 *
 *         gcc -std=c99 -Wall -Wpedantic -O2 hpp.c -o hpp -lm
 *
 * Run with
 *
 *         ./hpp [N [S]] input
 *
 * Where N=side of the domain (must be even), S=number of time steps.
 *
 *
 * ## Example
 *
 * ./hpp 1024 256 walls.in
 *
 *
 * ## To produce an animation
 *
 * Compile with -DDUMP_ALL:
 *
 *      gcc -std=c99 -Wall -Wpedantic -O2 -DDUMP_ALL hpp.c -o hpp -lm
 *
 * then:
 *
 *      ffmpeg -y -i "hpp%05d.pgm" -vcodec mpeg4 movie.avi
 *
 *
 * ## Scene description language
 *
 * All cells of the domain are initially EMPTY. All coordinates are
 * real numbers in [0, 1]; they are automatically scaled to the
 * resolution N used for the image.
*
 * c x y r t
 *
 *   Draw a circle centered ad (x, y) with radius r filled with
 *   particles of type t (0=WALL, 1=GAS, 2=EMPTY)
 *
 *
 * b x1 y1 x2 y2 t
 *
 *   Draw a rectangle with opposite corners (x1,y1) and (x2,y2) filled
 *   with particles of type t (0=WALL, 1=GAS, 2=EMPTY)
 *
 *
 * r x1 y1 x2 y2 p
 *
 *   Fill the rectangle with opposite corners (x1,y1), (x2,y2) with
 *   GAS particles with probability p \in [0, 1]. Only EMPTY cells
 *   might be filled with gas particles, everything else is not
 *   modified.
 *
 ****************************************************************************/
#include <stdio.h>
#include <stdlib.h>
#include <math.h> /* for ceil() */
#include <assert.h>
#include "hpc.h"

#define BLKDIM 32

typedef enum {
    WALL,
    GAS,
    EMPTY
} cell_value_t;

typedef enum {
    ODD_PHASE = -1,
    EVEN_PHASE = 1
} phase_t;

/* type of a cell of the domain */
typedef unsigned char cell_t;

/* Simplifies indexing on a N*N grid */
__host__ __device__ int IDX(int i, int j, int N)
{
    /* wrap-around */
    i = (i+N) % N;
    j = (j+N) % N;
    return i*N + j;
}

/* Swap the content of cells a and b, provided that neither is a WALL;
   otherwise, do nothing. */
__host__ __device__ void swap_cells(cell_t *a, cell_t *b)
{
    if ((*a != WALL) && (*b != WALL)) {
        cell_t tmp = *a;
        *a = *b;
        *b = tmp;
    }
}

/* Compute the `next` grid given the `cur`-rent configuration. */
void step( const cell_t *cur, cell_t *next, int N, phase_t phase )
{
    int i, j;

    assert(cur != NULL);
    assert(next != NULL);

    /* Loop over all coordinates (i,j) s.t. both i and j are even */
    for (i=0; i<N; i+=2) {
        for (j=0; j<N; j+=2) {
            /**
             * If phase==EVEN_PHASE:
             * ab
             * cd
             *
             * If phase==ODD_PHASE:
             * dc
             * ba
             */
            const int a = IDX(i      , j      , N);
            const int b = IDX(i      , j+phase, N);
            const int c = IDX(i+phase, j      , N);
            const int d = IDX(i+phase, j+phase, N);
            next[a] = cur[a];
            next[b] = cur[b];
            next[c] = cur[c];
            next[d] = cur[d];
            if ((((next[a] == EMPTY) != (next[b] == EMPTY)) &&
                 ((next[c] == EMPTY) != (next[d] == EMPTY))) ||
                (next[a] == WALL) || (next[b] == WALL) ||
                (next[c] == WALL) || (next[d] == WALL)) {
                swap_cells(&next[a], &next[b]);
                swap_cells(&next[c], &next[d]);
            } else {
                swap_cells(&next[a], &next[d]);
                swap_cells(&next[b], &next[c]);
            }
        }
    }
}

__global__ void cuda_step(cell_t *cur, cell_t *next, int N, phase_t phase) {

    const int i = (threadIdx.y + blockIdx.y * blockDim.y) * 2;
    const int j = (threadIdx.x + blockIdx.x * blockDim.x) * 2;

    assert(cur != NULL);
    assert(next != NULL);

    if (i < N && j < N) {
        const int a = IDX(i      , j      , N);
        const int b = IDX(i      , j+phase, N);
        const int c = IDX(i+phase, j      , N);
        const int d = IDX(i+phase, j+phase, N);
        next[a] = cur[a];
        next[b] = cur[b];
        next[c] = cur[c];
        next[d] = cur[d];
        if ((((next[a] == EMPTY) != (next[b] == EMPTY)) &&
                ((next[c] == EMPTY) != (next[d] == EMPTY))) ||
            (next[a] == WALL) || (next[b] == WALL) ||
            (next[c] == WALL) || (next[d] == WALL)) {
            swap_cells(&next[a], &next[b]);
            swap_cells(&next[c], &next[d]);
        } else {
            swap_cells(&next[a], &next[d]);
            swap_cells(&next[b], &next[c]);
        }
    }
}

/**
 ** The functions below are used to draw onto the grid; since they are
 ** called during initialization only, they do not need to be
 ** parallelized.
 **/
void box( cell_t *grid, int N, float x1, float y1, float x2, float y2, cell_value_t t )
{
    const int ix1 = ceil(fminf(x1, x2) * N);
    const int ix2 = ceil(fmaxf(x1, x2) * N);
    const int iy1 = ceil(fminf(y1, y1) * N);
    const int iy2 = ceil(fmaxf(y1, y2) * N);
    int i, j;
    for (i = iy1; i <= iy2; i++) {
        for (j = ix1; j <= ix2; j++) {
            const int ij = IDX(N-1-i, j, N);
            grid[ij] = t;
        }
    }
}

void circle( cell_t *grid, int N, float x, float y, float r, cell_value_t t )
{
    const int ix = ceil(x * N);
    const int iy = ceil(y * N);
    const int ir = ceil(r * N);
    int dx, dy;
    for (dy = -ir; dy <= ir; dy++) {
        for (dx = -ir; dx <= ir; dx++) {
            if (dx*dx + dy*dy <= ir*ir) {
                const int ij = IDX(N-1-iy-dy, ix+dx, N);
                grid[ij] = t;
            }
        }
    }
}

void random_fill( cell_t *grid, int N, float x1, float y1, float x2, float y2, float p )
{
    const int ix1 = ceil(fminf(x1, x2) * N);
    const int ix2 = ceil(fmaxf(x1, x2) * N);
    const int iy1 = ceil(fminf(y1, y1) * N);
    const int iy2 = ceil(fmaxf(y1, y2) * N);
    int i, j;
    for (i = iy1; i <= iy2; i++) {
        for (j = ix1; j <= ix2; j++) {
            const int ij = IDX(N-1-i, j, N);
            if (grid[ij] == EMPTY && ((float)rand())/RAND_MAX < p)
                grid[ij] = GAS;
        }
    }
}

void read_problem( FILE *filein, cell_t *grid, int N )
{
    int i,j;
    int nread;
    char op;

    for (i=0; i<N; i++) {
        for (j=0; j<N; j++) {
            const int ij = IDX(i,j,N);
            grid[ij] = EMPTY;
        }
    }

    while ((nread = fscanf(filein, " %c", &op)) == 1) {
        int t;
        float x1, y1, x2, y2, r, p;
        int retval;

        switch (op) {
        case 'c' : /* circle */
            retval = fscanf(filein, "%f %f %f %d", &x1, &y1, &r, &t);
            assert(retval == 4);
            circle(grid, N, x1, y1, r, (cell_value_t)t);
            break;
        case 'b': /* box */
            retval = fscanf(filein, "%f %f %f %f %d", &x1, &y1, &x2, &y2, &t);
            assert(retval == 5);
            box(grid, N, x1, y1, x2, y2, (cell_value_t)t);
            break;
        case 'r': /* random_fill */
            retval = fscanf(filein, "%f %f %f %f %f", &x1, &y1, &x2, &y2, &p);
            assert(retval == 5);
            random_fill(grid, N, x1, y1, x2, y2, p);
            break;
        default:
            fprintf(stderr, "FATAL: Unrecognized command `%c`\n", op);
            exit(EXIT_FAILURE);
        }
    }
}


/* Write an image of `grid` to a file in PGM (Portable Graymap)
   format. `frameno` is the time step number, used for labeling the
   output file. */
void write_image( const cell_t *grid, int N, int frameno )
{
    FILE *f;
    char fname[128];

    snprintf(fname, sizeof(fname), "hpp%05d.pgm", frameno);
    if ((f = fopen(fname, "w")) == NULL) {
        printf("Cannot open \"%s\" for writing\n", fname);
        abort();
    }
    fprintf(f, "P5\n");
    fprintf(f, "# produced by hpp\n");
    fprintf(f, "%d %d\n", N, N);
    fprintf(f, "%d\n", EMPTY); /* highest shade of grey (0=black) */
    fwrite(grid, 1, N*N, f);
    fclose(f);
}

int main( int argc, char* argv[] )
{
    int t, N, nsteps;
    FILE *filein;

    srand(1234); /* Initialize PRNG deterministically */

    if ( (argc < 2) || (argc > 4) ) {
        fprintf(stderr, "Usage: %s [N [S]] input\n", argv[0]);
        return EXIT_FAILURE;
    }

    if (argc > 2) {
        N = atoi(argv[1]);
    } else {
        N = 512;
    }

    if (argc > 3) {
        nsteps = atoi(argv[2]);
    } else {
        nsteps = 32;
    }

    if (N % 2 != 0) {
        fprintf(stderr, "FATAL: the domain size N must be even\n");
        return EXIT_FAILURE;
    }

    if ((filein = fopen(argv[argc-1], "r")) == NULL) {
        fprintf(stderr, "FATAL: can not open \"%s\" for reading\n", argv[argc-1]);
        return EXIT_FAILURE;
    }

    const size_t GRID_SIZE = N*N*sizeof(cell_t);
    cell_t *cur = (cell_t*)malloc(GRID_SIZE);
    assert(cur != NULL);
    cell_t *next = (cell_t*)malloc(GRID_SIZE);
    assert(next != NULL);

    read_problem(filein, cur, N);

    const dim3 gridSize ((N + BLKDIM - 1) / BLKDIM / 2, (N + BLKDIM - 1) / BLKDIM / 2);
    const dim3 blockSize (BLKDIM, BLKDIM);

    // create device variables and allocate CUDA memory
    cell_t *d_cur, *d_next;
    hipMalloc((void **)&d_cur, GRID_SIZE);
    hipMalloc((void **)&d_next, GRID_SIZE);
    hipMemcpy(d_cur, cur, GRID_SIZE, hipMemcpyHostToDevice);

    double start_time = hpc_gettime();
    for (t=0; t<nsteps; t++) {
        cuda_step<<<gridSize, blockSize>>>(d_cur, d_next, N, EVEN_PHASE);
        cudaCheckError();
        cuda_step<<<gridSize, blockSize>>>(d_next, d_cur, N, ODD_PHASE);
        cudaCheckError();
    }
    double finish_time = hpc_gettime();
    printf("Execution time: %fs\n", finish_time - start_time);

    hipMemcpy(cur, d_cur, GRID_SIZE, hipMemcpyDeviceToHost);
    // free cuda memory
    hipFree(d_cur);
    hipFree(d_next);
    write_image(cur, N, t);
    free(cur);
    free(next);
    fclose(filein);
    return EXIT_SUCCESS;
}
